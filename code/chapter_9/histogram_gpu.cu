#include "hip/hip_runtime.h"
#include "../common/book.h"
#include <time.h>  // 引入 time.h 头文件

#define SIZE (100 * 1024 * 1024)

__global__ void histo_kernel(unsigned char *buffer, long size, unsigned int *histo)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    
    while (i < size)
    {
        atomicAdd(&(histo[buffer[i]]), 1);
        i += stride;
    }
}

int main(void)
{

    unsigned char *buffer = (unsigned char*)big_random_block(SIZE);
    
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    // allocate memory on the GPU for the file's data
    unsigned char *dev_buffer;
    unsigned int *dev_histo;
    HANDLE_ERROR(hipMalloc((void**)&dev_buffer, SIZE));
    HANDLE_ERROR(hipMemcpy(dev_buffer, buffer, SIZE, hipMemcpyHostToDevice));

    HANDLE_ERROR(hipMalloc((void**)&dev_histo, 256 * sizeof(long)));
    HANDLE_ERROR(hipMemset(dev_histo, 0, 256 * sizeof(int)));

    // kernel code here
    hipDeviceProp_t prop;
    HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
    int blocks = prop.multiProcessorCount;
    histo_kernel<<<blocks * 2, 256>>>(dev_buffer, SIZE, dev_histo);

    unsigned int histo[256];
    HANDLE_ERROR(hipMemcpy(histo, dev_histo, 256 * sizeof(int), hipMemcpyDeviceToHost));

    // get stop time, and display the timing results
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    float elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Time to generate: %3.1f ms \n", elapsedTime);

    long histoCount = 0;
    for (int i = 0; i < 256; i ++ )
    {
        histoCount += histo[i];
    }
    printf("Histogram Sum: %ld\n", histoCount);

    // vefiry that we have the same counts via CPU
    for (int i = 0; i < SIZE; i ++ )
        histo[buffer[i]] --;
    for (int i = 0; i < 256; i ++ )
    {
        if (histo[i] != 0)
            printf("Failuer at %d!\n", i);
    }

    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));
    hipFree(dev_histo);
    hipFree(dev_buffer);
    free(buffer);

    return 0;
}
