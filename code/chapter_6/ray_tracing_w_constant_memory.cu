#include "hip/hip_runtime.h"
// This simplified ray tracing implementation is to understand how to use constant memory in GPU
#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "../common/cpu_bitmap_save.h"

#define INF 2e10f
#define DIM 512
#define rnd(x) (x * rand() / RAND_MAX)
#define SPHERES 20 // If you want to see preformance improvement using constant memory, you should adjust ther number based on you device.

struct Sphere
{
    float r, b, g;
    float radius;
    float x, y, z;

    __device__ float hit(float ox, float oy, float *n)
    {
        float dx = ox - x;
        float dy = oy - y;
        if (dx * dx + dy * dy < radius * radius)
        {
            float dz = sqrtf(radius * radius - dx * dx - dy * dy);
            *n = dz / sqrtf(radius * radius);
            return dz + z;
        }
        return -INF;
    }
};

__constant__ Sphere s[SPHERES];
__global__ void kernel(unsigned char *ptr)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int offset = x + y * gridDim.x * blockDim.x;
    float ox = (x - DIM / 2);
    float oy = (y - DIM / 2);
    
    float r = 0, g = 0, b = 0;
    float maxz = -INF;
    for (int i = 0; i < SPHERES; i ++ )
    {
        float n;
        float t = s[i].hit(ox, oy, &n);
        if (t > maxz)
        {
            maxz = t;
            float fscale = n;
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
        }
    }
    ptr[offset * 4 + 0] = (int)(r * 255);
    ptr[offset * 4 + 1] = (int)(g * 255);
    ptr[offset * 4 + 2] = (int)(b * 255);
    ptr[offset * 4 + 3] = 255;
}

int main(void)
{
    // capture the start time
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    CPUBitmap bitmap(DIM, DIM);
    unsigned char *dev_bitmap;

    // allocate memory on the GPU fot the output bitmap
    HANDLE_ERROR(hipMalloc((void**)&dev_bitmap, bitmap.image_size()));

    // allocate temop memory, initialize it, copy to constant
    // memory on the GPU, and then free our temp memory
    Sphere *temp_s = (Sphere*)malloc(sizeof(Sphere) * SPHERES);
    for (int i = 0; i < SPHERES; i ++ )
    {
        temp_s[i].r = rnd(1.0f);
        temp_s[i].g = rnd(1.0f);
        temp_s[i].b = rnd(1.0f);
        temp_s[i].x = rnd(1000.0f) - 500;
        temp_s[i].y = rnd(1000.0f) - 500;
        temp_s[i].z = rnd(1000.0f) - 500;
        temp_s[i].radius = rnd(100.0f) + 20;
    }
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(s), temp_s, sizeof(Sphere) * SPHERES));
    free(temp_s);

    // generate a bitmap for our sphere data
    dim3 grids(DIM / 16, DIM / 16);
    dim3 threads(16, 16);
    kernel<<<grids, threads>>>(dev_bitmap);

    HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));

    float elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

    printf("Time to generate: %3.1f ms\n", elapsedTime);
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    
    bitmap.display_and_exit();

    // free our memory
    hipFree(dev_bitmap);
}