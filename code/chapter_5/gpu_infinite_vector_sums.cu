#include "hip/hip_runtime.h"
#include "../common/book.h"

#define N (1024 * 1024 * 512)

__global__ void add(int *a, int *b, int *c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x; // handle the data at this index
    while (tid < N)
    {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int main(void)
{
    // int a[N], b[N], c[N];

    // Declare arrays on the host. Instead of using stack memory (which is limited),
    // allocate memory for arrays 'a', 'b', and 'c' on the heap to handle large sizes.
    int *a = (int *)malloc(N * sizeof(int));
    int *b = (int *)malloc(N * sizeof(int));
    int *c = (int *)malloc(N * sizeof(int));

    // Check if memory allocation on the host was successful
    if (!a || !b || !c) 
    {
        printf("Failed to allocate memory on host\n");
        return -1;
    }

    int *dev_a, *dev_b, *dev_c;

    // allocate the memory on the GPU
    HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

    // fill the arrays "a" and "b" on the CPU
    for (int i = 0; i < N; i ++ )
    {
        a[i] = -i;
        b[i] = i * i;
    }

    // copy the arrays "a" and "b" to the GPU
    HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

    add<<<1024, 1024>>>(dev_a, dev_b, dev_c);

    // copy the array "c" back from GPU to the CPU
    HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

    // vecrify that the GPU did the work we requested
    bool success = true;
    for (int i = 0; i < N; i ++ )
    {
        if ((a[i] + b[i] != c[i]))
        {
            printf("Error: %d + %d != %d\n", a[i], b[i], c[i]);
            success = false;
        }
    }
    if (success) printf("We did it\n");

    // Free the host memory allocated on the heap
    free(a);
    free(b);
    free(c);

    // free the memory allocated on the GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;

}