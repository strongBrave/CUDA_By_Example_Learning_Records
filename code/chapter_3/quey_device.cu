// #include "../book.h"

// int main(void) 
// {
//     hipDeviceProp_t prop;

//     int count;
//     HANDLE_ERROR( hipGetDeviceCount( &count));
//     for (int i = 0; i < count; i ++)
//     {
//         HANDLE_ERROR( hipGetDeviceProperties(&prop, i));

//         // Do something with our device's properties.
//         printf("  --- General Information for device %d --- \n", i);
//         printf("Name: %s\n", prop.name);
//         printf("Compute capability: %d.%d\n", prop.major, prop.minor);
//         printf("Clock rate: %d\n", prop.clockRate);
//         printf("Device copy overlap: ");
//         if (prop.deviceOverlap)
//             printf("Enabled\n");
//         else printf("Disbaled\n");
//         printf("Kernel exection timeout: ");
//         if (prop.kernelExecTimeoutEnabled) printf("Enabled\n");
//         else printf("Disabled\n");

//         printf("  --- Memory Information for device %d --- \n", i);
//         printf("Total global mem: %zu\n", prop.totalGlobalMem);
//         printf("Total constant Mem: %zu\n", prop.totalConstMem);
//         printf("Max mem pitch: %zu\n", prop.memPitch);
//         printf("Texture Alignment: %zu\n", prop.textureAlignment);

//         printf("  --- MP Information for device %d ---\n", i);
//         printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
//         printf("Shared mem per mp: %zu\n", prop.sharedMemPerBlock);
//         printf("Registers per mp: %d\n", prop.regsPerBlock);
//         printf("Threads in warp: %d\n", prop.warpSize);
//         printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
//         printf("Max thread dimensions: (%d, %d, %d)\n", 
//                 prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
//         printf("Max grid dimensions: (%d, %d, %d)\n",
//                 prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
//         printf("\n");
//     }
//     return 0;
// }


# include "../book.h"

int main(void)
{
    hipDeviceProp_t prop;
    int dev;

    HANDLE_ERROR(hipGetDevice(&dev));
    printf("ID of current CUDA device: %d\n", dev);

    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 1;
    prop.minor = 3;
    HANDLE_ERROR(hipChooseDevice(&dev, &prop));
    printf("ID of CUDA device closest to revision 1,3: %d\n", dev);
    HANDLE_ERROR(hipSetDevice(dev));

    return 0;
}