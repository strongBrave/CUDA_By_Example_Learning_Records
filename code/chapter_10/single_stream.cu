#include "hip/hip_runtime.h"
#include "../common/book.h"
#define N   (1024*1024)
#define FULL_DATA_SIZE   (N*20)

__global__ void kernel( int *a, int *b, int *c ) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N)
    {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f; 
        float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;         
        c[idx] = (as + bs) / 2;
    }
}

int main(void)
{
    hipDeviceProp_t prop;
    int whichDevice;
    HANDLE_ERROR(hipGetDevice(&whichDevice));
    HANDLE_ERROR(hipGetDeviceProperties(&prop, whichDevice));
    if (! prop.deviceOverlap)
    {
        printf("Device will not handle overlaps, so no"
        "speed up from streams\n");

        return 0;
    }

    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    // initialize the stream
    hipStream_t stream;
    HANDLE_ERROR(hipStreamCreate(&stream));

    int *host_a, *host_b, *host_c;
    int *dev_a, *dev_b, *dev_c;

    // allocate the memory on the GPU
    HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

    // allocate page-locked memory, used to stream
    HANDLE_ERROR(hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));

    for (int i = 0; i < FULL_DATA_SIZE; i ++ )
    {
        host_a[i] = rand();
        host_b[i] = rand();
    }
    
    // new loop over full data, in bite-sized chunks
    for (int i = 0; i < FULL_DATA_SIZE; i += N)
    {
        // copy the locked memory to the device, async
        HANDLE_ERROR(hipMemcpyAsync(dev_a, 
                                    host_a + i, 
                                    N * sizeof(int), 
                                    hipMemcpyHostToDevice, 
                                    stream));
        HANDLE_ERROR(hipMemcpyAsync(dev_b, 
                                    host_b + i, 
                                    N * sizeof(int),
                                    hipMemcpyHostToDevice,
                                    stream));
        kernel<<<N / 256, 256, 0, stream>>>(dev_a, dev_b, dev_c);

        // copy the data from device to locked memory
        HANDLE_ERROR(hipMemcpyAsync(host_c + i, 
                                    dev_c, 
                                    N * sizeof(int),
                                    hipMemcpyDeviceToHost,
                                    stream));
    }

    // copy result chunk from locked to full buffer
    HANDLE_ERROR(hipStreamSynchronize(stream));

    HANDLE_ERROR(hipEventRecord(stop, 0));

    HANDLE_ERROR(hipEventSynchronize(stop));
    float elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Time taken: %3.1f ms\n", elapsedTime);

    // cleanup the streams and memory
    HANDLE_ERROR(hipHostFree(host_a));
    HANDLE_ERROR(hipHostFree(host_b));
    HANDLE_ERROR(hipHostFree(host_c));
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c));

    HANDLE_ERROR(hipStreamDestroy(stream));
    
    return 0;
}